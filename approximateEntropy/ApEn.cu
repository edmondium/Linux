
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include <sstream>
__global__ void maxDistKernel(const double* data, int m, double r, double* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n - m + 1) {
        int count = 0;
        for (int j = 0; j < n - m + 1; ++j) {
            double max_dist = 0.0;
            for (int k = 0; k < m; ++k) {
                double dist = fabs(data[i + k] - data[j + k]);
                max_dist = fmax(max_dist, dist);
            }
            if (max_dist <= r) {
                ++count;
            }
        }
        C[i] = static_cast<double>(count) / (n - m + 1);
    }
}

double phi(int m, double r, const std::vector<double>& data) {
    int n = data.size();
    double* d_data;
    double* d_C;
    hipMalloc(&d_data, n * sizeof(double));
    hipMalloc(&d_C, (n - m + 1) * sizeof(double));
    hipMemcpy(d_data, data.data(), n * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n - m + 1 + blockSize - 1) / blockSize;
    maxDistKernel<<<numBlocks, blockSize>>>(d_data, m, r, d_C, n);
    hipDeviceSynchronize();

    std::vector<double> C(n - m + 1);
    hipMemcpy(C.data(), d_C, (n - m + 1) * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_C);

    double sum = 0.0;
    for (double c : C) {
        sum += std::log(c);
    }
    return sum / (n - m + 1);
}

double approximateEntropy(const std::vector<double>& time_series, int m, double r) {
    return std::abs(phi(m, r, time_series) - phi(m + 1, r, time_series));
}

std::vector<double> readDataFromFile(const std::string& filename) {
    std::vector<double> data;
    std::ifstream file(filename);
    if (file.is_open()) {
        std::string line;
        while (std::getline(file, line)) {
            std::istringstream iss(line);
            double value;
            while (iss >> value) {
                data.push_back(value);
            }
        }
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filename << std::endl;
    }
    return data;
}

int main() {
    std::string filename = "data.txt"; // Replace with your file name
    std::vector<double> data = readDataFromFile(filename);
    if (data.empty()) {
        std::cerr << "No data read from file." << std::endl;
        return 1;
    }

    int m;
    double r;
    std::cout << "Length of compared run of data m: ";
    std::cin >> m;
    std::cout << "Filtering level r: ";
    std::cin >> r;

    double apEn = approximateEntropy(data, m, r);
    std::cout << "Approximate Entropy: " << apEn << std::endl;

    return 0;
}
