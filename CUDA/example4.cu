// Floyd-Warshall algorithm

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
// Define the size of the graph
#define n 4

// Define the kernel function
__global__ void FW_APSP(int *D, int)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    for (int k = 0; k < n; k++)
    {
        if (D[i * n + j] > D[i * n + k] + D[k * n + j])
        {
            D[i * n + j] = D[i * n + k] + D[k * n + j];
        }
        __syncthreads();
    }
    
}

int main()
{
    // Define the graph as an adjacency matrix
    int h_D[n][n] = {{0, 9999, -2, 9999},
                     {4, 0, 3, 9999},
                     {9999, 9999, 0, 2},
                     {9999, -1, 9999, 0}};
    
    // Allocate memory for the array on the device
    int *d_D;
    hipMalloc(&d_D, n * n * sizeof(int));
    
    // Copy the data from the host to the device
    hipMemcpy(d_D, h_D, n * n * sizeof(int), hipMemcpyHostToDevice);
    
    // Define the block size
    dim3 threadsPerBlock(n, n);
    
    // Launch the kernel
    FW_APSP<<<1, threadsPerBlock>>>(d_D, n);
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        //printf("CUDA error: %s\n", cudaGetErrorString(err));
        cout << "CUDA error: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
    
    // Copy the data from the device to the host
    hipMemcpy(h_D, d_D, n * n * sizeof(int), hipMemcpyDeviceToHost);
    
    // Print the result
    //printf("The shortest paths are:\n");
    cout << "The shortest paths are:" << endl;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            //printf("%d ", h_D[i][j]);
            cout << " " << h_D[i][j];
        }
        //printf("\n");
        cout << endl;
    }
    
    // Free the memory on the device
    hipFree(d_D);
    
    return 0;
}