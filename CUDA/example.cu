
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}
int main(void)
{
    int a = 1, b = 2, c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    // Allocate space for device copies of a, b, c
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));
    // Copy inputs to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<1, 1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << "" << a << " + " << b << " = " << c << endl;
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}